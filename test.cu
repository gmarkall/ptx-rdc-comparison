
#include <hip/hip_runtime.h>
extern __device__ int g;
__device__ int arr[10] = { 0 };

// __device__ function not called within the translation unit
__device__ int f_unused(int* x, int y, int i)
{
  int j = blockIdx.x;
  return x[i] + y + g + arr[j];
}


// __device__ function called within the translation unit that can't be inlined
__device__ __noinline__ int f_out(int* x, int y, int i)
{
  int j = blockIdx.x;
  return x[i] + y + g + arr[j];
}


// __device__ function called within the translation unit that is inlined
__device__ __forceinline__ int f_in(int* x, int y, int i)
{
  int j = blockIdx.x;
  return x[i] + y + arr[j];
}


__global__ void f1(int* x, int* y)
{
  int i = threadIdx.x;
  y[i] = f_out(x, y[i], 2);
  y[i] += f_in(x, y[i], 2);
}
