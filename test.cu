
#include <hip/hip_runtime.h>
extern __device__ int g;
__device__ int arr[10] = { 0 };

__device__ int f(int* x, int y, int i)
{
  int j = blockIdx.x;
  return x[i] + y + g + arr[j];
}

__global__ void f1(int* x, int* y)
{
  int i = threadIdx.x;
  y[i] = f(x, y[i], 2);
}
